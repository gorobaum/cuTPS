#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_occupancy.h"

#include "cutps.h"

inline
hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        std::cout << "CUDA Runtime Error: \n" << hipGetErrorString(result) << std::endl;
        assert(result == hipSuccess);
    }
    return result;
}

// Kernel definition
__device__ short cudaGetPixel(int x, int y, int z, short* image, int width, int height, int slices) {
  if (x > width-1 || x < 0) return 0;
  if (y > height-1 || y < 0) return 0;
  if (z > slices-1 || z < 0) return 0;
  return image[z*height*width+x*height+y];
}

// Kernel definition
__device__ short cudaTrilinearInterpolation(float x, float y, float z, short* image, 
                                            int width, int height, int slices) {
  int u = trunc(x);
  int v = trunc(y);
  int w = trunc(z);

  float xd = (x - u);
  float yd = (y - v);
  float zd = (z - w);

  short c00 = cudaGetPixel(u, v, w, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v, w, image, width, height, slices)*xd;

  short c10 = cudaGetPixel(u, v+1, w, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v+1, w, image, width, height, slices)*xd;

  short c01 = cudaGetPixel(u, v, w+1, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v, w+1, image, width, height, slices)*xd;

  short c11 = cudaGetPixel(u, v+1, w+1, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v+1, w+1, image, width, height, slices)*xd;

  short c0 = c00*(1-yd)+c10*yd;
  short c1 = c01*(1-yd)+c11*yd;

  short result = c0*(1-zd)+c1*zd;
  if (result < 0) result = 0;
  return result;
}

// Kernel definition
__global__ void tpsCuda(short* cudaImage, short* cudaRegImage, float* solutionX, float* solutionY, 
                        float* solutionZ, int width, int height, int slices, float* keyX, float* keyY, 
                        float* keyZ, int numOfKeys) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  float newX = solutionX[0] + x*solutionX[1] + y*solutionX[2] + z*solutionX[3];
  float newY = solutionY[0] + x*solutionY[1] + y*solutionY[2] + z*solutionY[3];
  float newZ = solutionZ[0] + x*solutionZ[1] + y*solutionZ[2] + z*solutionZ[3];

  for (int i = 0; i < numOfKeys; i++) {
    float r = (x-keyX[i])*(x-keyX[i]) + (y-keyY[i])*(y-keyY[i]) + (z-keyZ[i])*(z-keyZ[i]);
    if (r != 0.0) {
      newX += r*log(r) * solutionX[i+4];
      newY += r*log(r) * solutionY[i+4];
      newZ += r*log(r) * solutionZ[i+4];
    }
  }
  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0)
        cudaRegImage[z*height*width+x*height+y] = cudaTrilinearInterpolation(newX, newY, newZ, cudaImage, width, height, slices);
}

void startTimeRecord(hipEvent_t *start, hipEvent_t *stop) {
  checkCuda(hipEventCreate(start));
  checkCuda(hipEventCreate(stop));
  checkCuda(hipEventRecord(*start, 0));
}

void showExecutionTime(hipEvent_t *start, hipEvent_t *stop, std::string output) {
  checkCuda(hipEventRecord(*stop, 0));
  checkCuda(hipEventSynchronize(*stop));
  float elapsedTime;
  checkCuda(hipEventElapsedTime(&elapsedTime, *start, *stop));
  checkCuda(hipEventDestroy(*start));
  checkCuda(hipEventDestroy(*stop));
  std::cout << output << elapsedTime << " ms\n";
}

short* runTPSCUDA(tps::CudaMemory cm, std::vector<int> dimensions, int numberOfCPs) {
  dim3 threadsPerBlock(8, 8, 8);
  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
                 std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
                 std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int slice = 0; slice < dimensions[2]; slice++)
    for (int col = 0; col < dimensions[0]; col++)
      for (int row = 0; row < dimensions[1]; row++)
        regImage[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCuda<<<numBlocks, threadsPerBlock>>>(cm.getTargetImage(), cm.getRegImage(), cm.getSolutionX(), cm.getSolutionY(), 
                                          cm.getSolutionZ(), dimensions[0], dimensions[1], dimensions[2], cm.getKeypointX(), 
                                          cm.getKeypointY(), cm.getKeypointZ(), numberOfCPs);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(regImage, cm.getRegImage(), dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short), hipMemcpyDeviceToHost));

  showExecutionTime(&start, &stop, "callKernel execution time = ");
  return regImage;
}