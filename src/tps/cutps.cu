#include "hip/hip_runtime.h"
#include <cassert>
#include <sstream>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_occupancy.h"

#include "cutps.h"

inline
hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        std::cout << "CUDA Runtime Error: \n" << hipGetErrorString(result) << std::endl;
        assert(result == hipSuccess);
    }
    return result;
}

// Kernel definition
__device__ short cudaGetPixel(int x, int y, int z, short* image, int width, int height, int slices) {
  if (x > width-1 || x < 0) return 0;
  if (y > height-1 || y < 0) return 0;
  if (z > slices-1 || z < 0) return 0;
  return image[z*height*width+y*width+x];
}

// Kernel definition
__device__ short cudaTrilinearInterpolation(float x, float y, float z, short* image,
                                            int width, int height, int slices) {
  int u = trunc(x);
  int v = trunc(y);
  int w = trunc(z);

  float xd = (x - u);
  float yd = (y - v);
  float zd = (z - w);

  short c00 = cudaGetPixel(u, v, w, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v, w, image, width, height, slices)*xd;

  short c10 = cudaGetPixel(u, v+1, w, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v+1, w, image, width, height, slices)*xd;

  short c01 = cudaGetPixel(u, v, w+1, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v, w+1, image, width, height, slices)*xd;

  short c11 = cudaGetPixel(u, v+1, w+1, image, width, height, slices)*(1-xd)
            + cudaGetPixel(u+1, v+1, w+1, image, width, height, slices)*xd;

  short c0 = c00*(1-yd)+c10*yd;
  short c1 = c01*(1-yd)+c11*yd;

  short result = c0*(1-zd)+c1*zd;
  if (result < 0) result = 0;
  return result;
}

// Kernel definition
__global__ void tpsCuda(short* cudaImage, short* cudaRegImage, float* solutionX, float* solutionY,
                        float* solutionZ, int width, int height, int slices, float* keyX, float* keyY,
                        float* keyZ, int numOfKeys) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  float newX = solutionX[0] + x*solutionX[1] + y*solutionX[2] + z*solutionX[3];
  float newY = solutionY[0] + x*solutionY[1] + y*solutionY[2] + z*solutionY[3];
  float newZ = solutionZ[0] + x*solutionZ[1] + y*solutionZ[2] + z*solutionZ[3];

  for (int i = 0; i < numOfKeys; i++) {
    float r = (x-keyX[i])*(x-keyX[i]) + (y-keyY[i])*(y-keyY[i]) + (z-keyZ[i])*(z-keyZ[i]);
    if (r != 0.0) {
      newX += r*log(r) * solutionX[i+4];
      newY += r*log(r) * solutionY[i+4];
      newZ += r*log(r) * solutionZ[i+4];
    }
  }

  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0)
        cudaRegImage[z*height*width+y*width+x] = cudaTrilinearInterpolation(newX, newY, newZ, cudaImage, width, height, slices);
}

// Kernel definition
__global__ void tpsCudaWithText(hipTextureObject_t textObj, short* cudaRegImage, float* solutionX, float* solutionY,
                        float* solutionZ, int width, int height, int slices, float* keyX, float* keyY,
                        float* keyZ, int numOfKeys) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  float newX = solutionX[0] + x*solutionX[1] + y*solutionX[2] + z*solutionX[3];
  float newY = solutionY[0] + x*solutionY[1] + y*solutionY[2] + z*solutionY[3];
  float newZ = solutionZ[0] + x*solutionZ[1] + y*solutionZ[2] + z*solutionZ[3];

  for (int i = 0; i < numOfKeys; i++) {
    float r = (x-keyX[i])*(x-keyX[i]) + (y-keyY[i])*(y-keyY[i]) + (z-keyZ[i])*(z-keyZ[i]);
    if (r != 0.0) {
      newX += r*log(r) * solutionX[i+4];
      newY += r*log(r) * solutionY[i+4];
      newZ += r*log(r) * solutionZ[i+4];
    }
  }

  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0)
        cudaRegImage[z*width*height+y*width+x] = (short)tex3D<float>(textObj, newX, newY, newZ);
}

void startTimeRecord(hipEvent_t *start, hipEvent_t *stop) {
  checkCuda(hipEventCreate(start));
  checkCuda(hipEventCreate(stop));
  checkCuda(hipEventRecord(*start, 0));
}

void showExecutionTime(hipEvent_t *start, hipEvent_t *stop, std::string output) {
  checkCuda(hipEventRecord(*stop, 0));
  checkCuda(hipEventSynchronize(*stop));
  float elapsedTime;
  checkCuda(hipEventElapsedTime(&elapsedTime, *start, *stop));
  checkCuda(hipEventDestroy(*start));
  checkCuda(hipEventDestroy(*stop));
  std::cout << output << elapsedTime << " ms\n";
}

int getBlockSize(int maxBlockSize) {
  int maxOccupancyBlockSize = 0;
  float maxOccupancy = 0.0;
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);

  for (int blockSize = 32; blockSize <= maxBlockSize; blockSize += 32) {
    int numBlocks;        // Occupancy in terms of active blocks

    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocks,
        tpsCuda,
        blockSize,
        0);

    int activeWarps = numBlocks * blockSize / prop.warpSize;
    int maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
    float currentOccupancy = 1.0*activeWarps/maxWarps;
    if (currentOccupancy >= maxOccupancy ) {
      maxOccupancy = currentOccupancy;
      maxOccupancyBlockSize = blockSize;
    }
  }
  return maxOccupancyBlockSize;
}

dim3 calculateBestThreadsPerBlock(int blockSize, bool twoDim) {
  dim3 threadsPerBlock;
  std::vector<int> threadsPerDim(3, 1);
  int divisor = 8;
  int imageDimension;
  if (twoDim) {
    imageDimension = 2;
  } else {
    imageDimension = 3;
  }

  for (int i = 0; divisor > 1;) {
    if (blockSize%divisor == 0) {
      threadsPerDim[i%imageDimension] *= divisor;
      blockSize /= divisor;
      i++;
    } else {
      divisor /= 2;
    }
  }

  threadsPerBlock.x = threadsPerDim[0];
  threadsPerBlock.y = threadsPerDim[1];
  threadsPerBlock.z = threadsPerDim[2];

  return threadsPerBlock;
}

short* runTPSCUDA(tps::CudaMemory cm, std::vector<int> dimensions, int numberOfCPs, bool occupancy, bool twoDim, int blockSize) {
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }
  std::cout << "threadsPerBlock.x = " << threadsPerBlock.x << std::endl;
  std::cout << "threadsPerBlock.y = " << threadsPerBlock.y << std::endl;
  std::cout << "threadsPerBlock.z = " << threadsPerBlock.z << std::endl;

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
                 std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
                 std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int slice = 0; slice < dimensions[2]; slice++)
    for (int col = 0; col < dimensions[0]; col++)
      for (int row = 0; row < dimensions[1]; row++)
        regImage[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCuda<<<numBlocks, threadsPerBlock>>>(cm.getTargetImage(), cm.getRegImage(), cm.getSolutionX(), cm.getSolutionY(),
                                          cm.getSolutionZ(), dimensions[0], dimensions[1], dimensions[2], cm.getKeypointX(),
                                          cm.getKeypointY(), cm.getKeypointZ(), numberOfCPs);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(regImage, cm.getRegImage(), dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short), hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());
  return regImage;
}

short* runTPSCUDAWithText(tps::CudaMemory cm, std::vector<int> dimensions, int numberOfCPs, bool occupancy, bool twoDim, int blockSize) {
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }
  std::cout << "threadsPerBlock.x = " << threadsPerBlock.x << std::endl;
  std::cout << "threadsPerBlock.y = " << threadsPerBlock.y << std::endl;
  std::cout << "threadsPerBlock.z = " << threadsPerBlock.z << std::endl;

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
                 std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
                 std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int slice = 0; slice < dimensions[2]; slice++)
    for (int col = 0; col < dimensions[0]; col++)
      for (int row = 0; row < dimensions[1]; row++)
        regImage[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCudaWithText<<<numBlocks, threadsPerBlock>>>(cm.getTexObj(), cm.getRegImage(), cm.getSolutionX(), cm.getSolutionY(),
                                          cm.getSolutionZ(), dimensions[0], dimensions[1], dimensions[2], cm.getKeypointX(),
                                          cm.getKeypointY(), cm.getKeypointZ(), numberOfCPs);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(regImage, cm.getRegImage(), dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short), hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());
  return regImage;
}
