#include "hip/hip_runtime.h"
#include <cassert>
#include <sstream>
#include <iostream>

#include <armadillo>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_occupancy.h"

#include "cutps.h"

#include <cmath>

typedef struct {
  float x, y, z;
} Point;

hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    std::cout << "CUDA Runtime Error: \n" << hipGetErrorString(result) << std::endl;
    assert(result == hipSuccess);
  }
  return result;
}

void startTimeRecord(hipEvent_t *start, hipEvent_t *stop) {
  checkCuda(hipEventCreate(start));
  checkCuda(hipEventCreate(stop));
  checkCuda(hipEventRecord(*start, 0));
}

void showExecutionTime(hipEvent_t *start, hipEvent_t *stop, std::string output) {
  checkCuda(hipEventRecord(*stop, 0));
  checkCuda(hipEventSynchronize(*stop));
  float elapsedTime;
  checkCuda(hipEventElapsedTime(&elapsedTime, *start, *stop));
  checkCuda(hipEventDestroy(*start));
  checkCuda(hipEventDestroy(*stop));
  std::cout << output << elapsedTime/1000 << " s\n";
}


//=====================
//KERNEL
//====================


// Kernel definition
__device__ short cudaGetPixel(int x, int y, int z, short* image, int width, int height, int slices){
  if (x > width-1 || x < 0) return 0;
  if (y > height-1 || y < 0) return 0;
  if (z > slices-1 || z < 0) return 0;
  return image[z*height*width+y*width+x];
}

// Kernel definition
__device__ short cudaTrilinearInterpolation(float x, float y, float z, short* image,
    int width, int height, int slices) {
  int u = trunc(x);
  int v = trunc(y);
  int w = trunc(z);

  float xd = (x - u);
  float yd = (y - v);
  float zd = (z - w);

  short c00 = cudaGetPixel(u, v, w, image, width, height, slices)*(1-xd)
    + cudaGetPixel(u+1, v, w, image, width, height, slices)*xd;

  short c10 = cudaGetPixel(u, v+1, w, image, width, height, slices)*(1-xd)
    + cudaGetPixel(u+1, v+1, w, image, width, height, slices)*xd;

  short c01 = cudaGetPixel(u, v, w+1, image, width, height, slices)*(1-xd)
    + cudaGetPixel(u+1, v, w+1, image, width, height, slices)*xd;

  short c11 = cudaGetPixel(u, v+1, w+1, image, width, height, slices)*(1-xd)
    + cudaGetPixel(u+1, v+1, w+1, image, width, height, slices)*xd;

  short c0 = c00*(1-yd)+c10*yd;
  short c1 = c01*(1-yd)+c11*yd;

  short result = c0*(1-zd)+c1*zd;
  if (result < 0) result = 0;
  return result;
}

__device__ Point calculateNewPoint(float* solutionX, float* solutionY,
                            float* solutionZ, float* keyX, float* keyY,
                            float* keyZ, int x, int y, int z, int numOfKeys) {
  Point newPoint;

  newPoint.x = solutionX[0] + x*solutionX[1] + y*solutionX[2] + z*solutionX[3];
  newPoint.y = solutionY[0] + x*solutionY[1] + y*solutionY[2] + z*solutionY[3];
  newPoint.z = solutionZ[0] + x*solutionZ[1] + y*solutionZ[2] + z*solutionZ[3];

  for (int i = 0; i < numOfKeys; i++) {
    float r = (x-keyX[i])*(x-keyX[i]) + (y-keyY[i])*(y-keyY[i]) + (z-keyZ[i])*(z-keyZ[i]);
    if (r != 0.0) {
      newPoint.x += r*log(r) * solutionX[i+4];
      newPoint.y += r*log(r) * solutionY[i+4];
      newPoint.z += r*log(r) * solutionZ[i+4];
    }
  }

  return newPoint;
}

// Kernel definition
__global__ void tpsCuda(short* cudaImage, short* cudaRegImage, float* solutionX, float* solutionY,
    float* solutionZ, int width, int height, int slices, float* keyX, float* keyY,
    float* keyZ, int numOfKeys) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  Point newPoint = calculateNewPoint(solutionX, solutionY, solutionZ, keyX,
                                     keyY, keyZ, x, y, z, numOfKeys);

  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0)
        cudaRegImage[z*height*width+y*width+x] =
              cudaTrilinearInterpolation(newPoint.x, newPoint.y, newPoint.z,
                                         cudaImage, width, height, slices);
}

// Kernel definition
__global__ void tpsCudaWithText(hipTextureObject_t textObj, short* cudaRegImage, float* solutionX, float* solutionY,
    float* solutionZ, int width, int height, int slices, float* keyX, float* keyY,
    float* keyZ, int numOfKeys) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  Point newPoint = calculateNewPoint(solutionX, solutionY, solutionZ,
                                     keyX, keyY, keyZ, x, y, z, numOfKeys);

  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0)
        cudaRegImage[z*width*height+y*width+x] =
            (short)tex3D<float>(textObj, newPoint.x, newPoint.y, newPoint.z);
}

__global__ void tpsCudaWithoutInterpolation(float* cudapointsx,
    float* cudapointsy, float* cudapointsz, float* solutionx,
    float* solutiony, float* solutionz, int width, int height,
    int slices, float* keyx, float* keyy, float* keyz,
    int numofkeys) {

  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  Point newPoint = calculateNewPoint(solutionx, solutiony, solutionz,
                                     keyx, keyy, keyz, x, y, z, numofkeys);

  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0) {
        cudapointsx[z*height*width+y*width+x] = newPoint.x;
        cudapointsy[z*height*width+y*width+x] = newPoint.y;
        cudapointsz[z*height*width+y*width+x] = newPoint.z;
      }
}

__global__ void tpscudaVectorField(float* cudapointsx, float* cudapointsy, float* cudapointsz, float* solutionx, float* solutiony,
    float* solutionz, int width, int height, int slices, float* keyx, float* keyy,
    float* keyz, int numofkeys) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  int z = blockDim.z*blockIdx.z + threadIdx.z;

  Point newPoint = calculateNewPoint(solutionx, solutiony, solutionz,
                                     keyx, keyy, keyz, x, y, z, numofkeys);

  if (x <= width-1 && x >= 0)
    if (y <= height-1 && y >= 0)
      if (z <= slices-1 && z >= 0) {
        cudapointsx[z*height*width+y*width+x] = newPoint.x - x;
        cudapointsy[z*height*width+y*width+x] = newPoint.y - y;
        cudapointsz[z*height*width+y*width+x] = newPoint.z - z;
      }
}

short getPixel(int x, int y, int z, short* image, std::vector<int> dimensions) {
  if (x > dimensions[0]-1 || x < 0) return 0;
  if (y > dimensions[1]-1 || y < 0) return 0;
  if (z > dimensions[2]-1 || z < 0) return 0;
  return image[z*dimensions[0]*dimensions[1]+y*dimensions[0]+x];
}

short trilinearInterpolation(float x, float y, float z, short* image, std::vector<int> dimensions) {
  int u = trunc(x);
  int v = trunc(y);
  int w = trunc(z);

  float xd = (x - u);
  float yd = (y - v);
  float zd = (z - w);

  short c00 = getPixel(u, v, w, image, dimensions)*(1-xd)
    + getPixel(u+1, v, w, image, dimensions)*xd;

  short c10 = getPixel(u, v+1, w, image, dimensions)*(1-xd)
    + getPixel(u+1, v+1, w, image, dimensions)*xd;

  short c01 = getPixel(u, v, w+1, image, dimensions)*(1-xd)
    + getPixel(u+1, v, w+1, image, dimensions)*xd;

  short c11 = getPixel(u, v+1, w+1, image, dimensions)*(1-xd)
    + getPixel(u+1, v+1, w+1, image, dimensions)*xd;

  short c0 = c00*(1-yd)+c10*yd;
  short c1 = c01*(1-yd)+c11*yd;

  short result = c0*(1-zd)+c1*zd;
  if (result < 0) result = 0;
  return result;
}

int getBlockSize(int maxBlockSize) {
  int maxOccupancyBlockSize = 0;
  float maxOccupancy = 0.0;
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);

  for (int blockSize = 32; blockSize <= maxBlockSize; blockSize += 32) {
    int numBlocks;        // Occupancy in terms of active blocks

    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocks,
        tpsCuda,
        blockSize,
        0);

    int activeWarps = numBlocks * blockSize / prop.warpSize;
    int maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
    float currentOccupancy = 1.0*activeWarps/maxWarps;
    if (currentOccupancy >= maxOccupancy ) {
      maxOccupancy = currentOccupancy;
      maxOccupancyBlockSize = blockSize;
    }
  }
  return maxOccupancyBlockSize;
}

dim3 calculateBestThreadsPerBlock(int blockSize, bool twoDim) {
  dim3 threadsPerBlock;
  std::vector<int> threadsPerDim(3, 1);
  int divisor = 8;
  int imageDimension;
  if (twoDim) {
    imageDimension = 2;
  } else {
    imageDimension = 3;
  }

  for (int i = 0; divisor > 1;) {
    if (blockSize%divisor == 0) {
      threadsPerDim[i%imageDimension] *= divisor;
      blockSize /= divisor;
      i++;
    } else {
      divisor /= 2;
    }
  }

  threadsPerBlock.x = threadsPerDim[0];
  threadsPerBlock.y = threadsPerDim[1];
  threadsPerBlock.z = threadsPerDim[2];

  return threadsPerBlock;
}

short* runTPSCUDA(tps::CudaMemory cm, std::vector<int> dimensions, int numberOfCPs, bool occupancy, bool twoDim, int blockSize) {
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }
  std::cout << "threadsPerBlock.x = " << threadsPerBlock.x << std::endl;
  std::cout << "threadsPerBlock.y = " << threadsPerBlock.y << std::endl;
  std::cout << "threadsPerBlock.z = " << threadsPerBlock.z << std::endl;

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
      std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
      std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int slice = 0; slice < dimensions[2]; slice++)
    for (int col = 0; col < dimensions[0]; col++)
      for (int row = 0; row < dimensions[1]; row++)
        regImage[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCuda<<<numBlocks, threadsPerBlock>>>(cm.getTargetImage(), cm.getRegImage(), cm.getSolutionX(), cm.getSolutionY(),
      cm.getSolutionZ(), dimensions[0], dimensions[1], dimensions[2], cm.getKeypointX(),
      cm.getKeypointY(), cm.getKeypointZ(), numberOfCPs);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(regImage, cm.getRegImage(), dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short), hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());
  return regImage;
}

short* runTPSCUDAWithText(tps::CudaMemory cm, std::vector<int> dimensions, int numberOfCPs, bool occupancy, bool twoDim, int blockSize) {
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }
  std::cout << "threadsPerBlock.x = " << threadsPerBlock.x << std::endl;
  std::cout << "threadsPerBlock.y = " << threadsPerBlock.y << std::endl;
  std::cout << "threadsPerBlock.z = " << threadsPerBlock.z << std::endl;

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
      std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
      std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int slice = 0; slice < dimensions[2]; slice++)
    for (int col = 0; col < dimensions[0]; col++)
      for (int row = 0; row < dimensions[1]; row++)
        regImage[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCudaWithText<<<numBlocks, threadsPerBlock>>>(cm.getTexObj(), cm.getRegImage(), cm.getSolutionX(), cm.getSolutionY(),
      cm.getSolutionZ(), dimensions[0], dimensions[1], dimensions[2], cm.getKeypointX(),
      cm.getKeypointY(), cm.getKeypointZ(), numberOfCPs);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(regImage, cm.getRegImage(), dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short), hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());
  return regImage;
}

short* interpolateImage(short* imageVoxels, float* imagePointsX, float* imagePointsY, float* imagePointsZ, std::vector<int> dimensions) {
  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int x = 0; x < dimensions[0]; x++)
    for (int y = 0; y < dimensions[1]; y++)
      for (int z = 0; z < dimensions[2]; z++) {
        float newX = imagePointsX[z*dimensions[0]*dimensions[1]+y*dimensions[0]+x];
        float newY = imagePointsY[z*dimensions[0]*dimensions[1]+y*dimensions[0]+x];
        float newZ = imagePointsZ[z*dimensions[0]*dimensions[1]+y*dimensions[0]+x];
        short newValue = trilinearInterpolation(newX, newY, newZ, imageVoxels, dimensions);
        regImage[z*dimensions[0]*dimensions[1]+y*dimensions[0]+x] = newValue;
      }

  return regImage;
}

short* runTPSCUDAWithoutInterpolation(tps::CudaMemory cm, short* imageVoxels,
                    std::vector<int> dimensions, int numberOfCPs,
                    bool occupancy, bool twoDim, int blockSize) {
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }
  std::cout << "threadsPerBlock.x = " << threadsPerBlock.x << std::endl;
  std::cout << "threadsPerBlock.y = " << threadsPerBlock.y << std::endl;
  std::cout << "threadsPerBlock.z = " << threadsPerBlock.z << std::endl;

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
      std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
      std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  float* imagePointsX = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));
  float* imagePointsY = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));
  float* imagePointsZ = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCudaWithoutInterpolation<<<numBlocks, threadsPerBlock>>>(
      cm.getImagePointsX(), cm.getImagePointsY(), cm.getImagePointsZ(),
      cm.getSolutionX(), cm.getSolutionY(),cm.getSolutionZ(), dimensions[0],
      dimensions[1], dimensions[2], cm.getKeypointX(), cm.getKeypointY(),
      cm.getKeypointZ(), numberOfCPs);

  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(imagePointsX, cm.getImagePointsX(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(imagePointsY, cm.getImagePointsY(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(imagePointsZ, cm.getImagePointsZ(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());

  arma::wall_clock timer;
  timer.tic();
  short* regImage = interpolateImage(imageVoxels, imagePointsX, imagePointsY, imagePointsZ, dimensions);
  double time = timer.toc();
  std::cout << "Interpolation execution time(" << numberOfCPs << "): " << time << "s" << std::endl;

  return regImage;
}

float* generateDeforVectorAt(int x, int y, int z) {
  float* vectorField = (float*)std::malloc(3*sizeof(float));

  vectorField[0] = x + 2.0*sin(y/8.0) - 2.0*cos(z/16.0);
  vectorField[1] = y + 4.0*sin(x/8.0) - 2.0*sin(z/8.0);
  vectorField[2] = z + 2.0*sin(x/16.0) - 4.0*cos(y/8.0);

  return vectorField;
}

float normOf(float x, float y, float z) {
  return std::sqrt(x*x + y*y + z*z);
}

float calculateSD(float meanError, float *vectorFieldX, float* vectorFieldY,
    float* vectorFieldZ, std::vector<int> dimensions) {
  float standardDeviation = 0.0;

  for (int x = 0; x < dimensions[0]; x++)
    for (int y = 0; y < dimensions[1]; y++)
      for (int z = 0; z < dimensions[2]; z++) {
        float* evf = generateDeforVectorAt(x, y, z); // expectedVectorField
        int pos = z*dimensions[1]*dimensions[0]+x*dimensions[1]+y;
        float currentError = normOf(evf[0] - vectorFieldX[pos], evf[1] - vectorFieldY[pos],
            evf[2] - vectorFieldZ[pos])/normOf(evf[0], evf[1], evf[2]);
        standardDeviation += std::pow(currentError - meanError, 2);
      }
  standardDeviation /= (dimensions[0]*dimensions[1]*dimensions[2]*1.0);
  standardDeviation = std::sqrt(standardDeviation);

  return standardDeviation;
}

float calculateError(float *vectorFieldX, float* vectorFieldY,
    float* vectorFieldZ, std::vector<int> dimensions) {
  float error = 0.0;

  for (int x = 0; x < dimensions[0]; x++)
    for (int y = 0; y < dimensions[1]; y++)
      for (int z = 0; z < dimensions[2]; z++) {
        float* evf = generateDeforVectorAt(x, y, z); // expectedVectorField
        int pos = z*dimensions[1]*dimensions[0]+x*dimensions[1]+y;
        error += normOf(evf[0] - vectorFieldX[pos], evf[1] - vectorFieldY[pos],
            evf[2] - vectorFieldZ[pos])/normOf(evf[0], evf[1], evf[2]);
      }
  error /= (dimensions[0]*dimensions[1]*dimensions[2]*1.0);

  return error;
}

short* runTPSCUDAVectorFieldTest(tps::CudaMemory cm, short* imageVoxels,
    std::vector<int> dimensions,
    int numberOfCPs, bool occupancy,
    bool twoDim, int blockSize) {
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }
  std::cout << "threadsPerBlock.x = " << threadsPerBlock.x << std::endl;
  std::cout << "threadsPerBlock.y = " << threadsPerBlock.y << std::endl;
  std::cout << "threadsPerBlock.z = " << threadsPerBlock.z << std::endl;

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
      std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
      std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  float* vectorFieldX = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));
  float* vectorFieldY = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));
  float* vectorFieldZ = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));

  for (int slice = 0; slice < dimensions[2]; slice++)
    for (int col = 0; col < dimensions[0]; col++)
      for (int row = 0; row < dimensions[1]; row++) {
        vectorFieldX[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;
        vectorFieldY[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;
        vectorFieldZ[slice*dimensions[1]*dimensions[0]+col*dimensions[1]+row] = 0;
      }

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCudaWithoutInterpolation<<<numBlocks, threadsPerBlock>>>(
      cm.getImagePointsX(), cm.getImagePointsY(), cm.getImagePointsZ(),
      cm.getSolutionX(), cm.getSolutionY(),cm.getSolutionZ(), dimensions[0],
      dimensions[1], dimensions[2], cm.getKeypointX(), cm.getKeypointY(),
      cm.getKeypointZ(), numberOfCPs);

  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(vectorFieldX, cm.getImagePointsX(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(vectorFieldY, cm.getImagePointsY(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(vectorFieldZ, cm.getImagePointsZ(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());

  arma::wall_clock timer;
  timer.tic();
  float error = calculateError(vectorFieldX, vectorFieldY, vectorFieldZ, dimensions);
  float sd = calculateSD(error, vectorFieldX, vectorFieldY, vectorFieldZ, dimensions);
  double time = timer.toc();
  std::cout << "Calculate error execution time(" << numberOfCPs << "): " << time << "s" << std::endl;
  std::cout << "Error for (" << numberOfCPs << ") = " << error << std::endl;
  std::cout << "SD for (" << numberOfCPs << ") = " << sd << std::endl;

  short* regImage = (short*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(short));

  for (int x = 0; x < dimensions[0]; x++)
    for (int y = 0; y < dimensions[1]; y++)
      for (int z = 0; z < dimensions[2]; z++) {
        regImage[z*dimensions[0]*dimensions[1]+y*dimensions[0]+x] = 0;
      }

  return regImage;
}

float radialDiff(tps::Image imageA, std::vector<std::vector<float> > keypointsA,
    tps::Image imageB, std::vector<std::vector<float> > keypointsB) {
  float diff = 0.0;
  int r = 3;
  for (int i = 0; i < keypointsA.size(); i++) {
    float radialA = imageA.radialSum(r, keypointsA[i]);
    float radialB = imageB.radialSum(r, keypointsB[i]);
    diff += (radialA-radialB);
  }
  diff /= keypointsA.size();
  return diff;
}

std::vector<std::vector<float> > applyVectorField(
    std::vector<std::vector<float> > keypoints, float* vectorFieldX,
    float* vectorFieldY, float* vectorFieldZ, std::vector<int> dimensions) {
  std::vector<std::vector<float> > newKeypoints;

  for (int i = 0; i < keypoints.size(); i++) {
    int x = keypoints[i][0];
    int y = keypoints[i][1];
    int z = keypoints[i][2];
    int pos = z*dimensions[0]*dimensions[1]+y*dimensions[0]+z;
    std::vector<float> newKeypoint;
    newKeypoint.push_back(keypoints[i][0] + vectorFieldX[pos]);
    newKeypoint.push_back(keypoints[i][1] + vectorFieldY[pos]);
    newKeypoint.push_back(keypoints[i][2] + vectorFieldZ[pos]);

    newKeypoints.push_back(newKeypoint);
  }

  return newKeypoints;
}

void runTPSRadialDiff(tps::CudaMemory cm, tps::Image regImage,
    tps::Image referenceImage, std::vector<std::vector<float> >referenceKeypoints,
    tps::Image targetImage, std::vector<std::vector<float> > targetKeypoints,
    bool occupancy, bool twoDim, int blockSize) {
  std::vector<int> dimensions = referenceImage.getDimensions();
  int numberOfCPs = targetKeypoints.size();
  dim3 threadsPerBlock;

  if (occupancy) {
    int maxBlockSize = getBlockSize(blockSize);
    threadsPerBlock = calculateBestThreadsPerBlock(maxBlockSize, twoDim);
  } else {
    threadsPerBlock.x = 8;
    threadsPerBlock.y = 8;
    if (twoDim) {
      threadsPerBlock.z = 1;
    } else {
      threadsPerBlock.z = 8;
    }
  }

  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x),
      std::ceil(1.0*dimensions[1]/threadsPerBlock.y),
      std::ceil(1.0*dimensions[2]/threadsPerBlock.z));

  float* vectorFieldX = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));
  float* vectorFieldY = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));
  float* vectorFieldZ = (float*)malloc(dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float));

  hipEvent_t start, stop;
  startTimeRecord(&start, &stop);

  tpsCudaWithoutInterpolation<<<numBlocks, threadsPerBlock>>>(
      cm.getImagePointsX(), cm.getImagePointsY(), cm.getImagePointsZ(),
      cm.getSolutionX(), cm.getSolutionY(),cm.getSolutionZ(), dimensions[0],
      dimensions[1], dimensions[2], cm.getKeypointX(), cm.getKeypointY(),
      cm.getKeypointZ(), numberOfCPs);

  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(vectorFieldX, cm.getImagePointsX(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(vectorFieldY, cm.getImagePointsY(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(vectorFieldZ, cm.getImagePointsZ(),
        dimensions[0]*dimensions[1]*dimensions[2]*sizeof(float),
        hipMemcpyDeviceToHost));

  std::ostringstream oss;
  oss << "callKernel execution time with sysDim(" << numberOfCPs << ")= ";

  showExecutionTime(&start, &stop, oss.str());

  arma::wall_clock timer;
  timer.tic();
  std::vector<std::vector<float> > resultKeypoints =
    applyVectorField(targetKeypoints, vectorFieldX, vectorFieldY,
                     vectorFieldZ, dimensions);

  float diffRefTar = radialDiff(referenceImage, referenceKeypoints,
                                targetImage, targetKeypoints);
  float diffRefRes = radialDiff(referenceImage, referenceKeypoints,
                                regImage, resultKeypoints);
  double time = timer.toc();
  std::cout << "Calculate error execution time(" << numberOfCPs << "): " << time << "s" << std::endl;
  std::cout << "Radial diff Ref Tar (" << numberOfCPs << ") = " << diffRefTar << std::endl;
  std::cout << "Radial diff Ref Reg (" << numberOfCPs << ") = " << diffRefRes << std::endl;


}

